#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#define MAXN 8200
#define MAXC 257
using namespace std;
__global__ void fun(int *n,int *m,int source[],int dest[],int *s,int *row,int *col,unsigned dom[],bool *f)
{
    int curr=blockIdx.x*blockDim.x+threadIdx.x;
   if(curr<(*m)&&dest[curr]!=(*s))
   {
    unsigned temp[MAXC],var[MAXC];
    for(int i=0;i<(*col);i++)
    	temp[i]=dom[dest[curr]*(*col)+i];
    for(int i=0;i<(*col);i++)
    	var[i]=temp[i]&dom[source[curr]*(*col)+i];
    var[dest[curr]>>5]|=(1<<(dest[curr]&31));
    if(*f)
    {
    	for(int i=0;i<(*col);i++)
    	{
    		if(temp[i]!=var[i])
    	    {
    	    	*f=false;
    	    	break;
    	    }
    	}
    }
    //__syncthreads();
    for(int i=0;i<(*col);i++)
    	atomicAnd(&dom[dest[curr]*(*col)+i],var[i]);
   }
}
int main()
{
	freopen("input.txt","r",stdin);
  freopen("output.txt","w",stdout);
	int hn,hm;
	scanf("%d%d",&hn,&hm);
	int *dn,*dm;
	hipMalloc((void**)&dn,sizeof(int));
	hipMalloc((void**)&dm,sizeof(int));
	hipMemcpy(dn,&hn,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dm,&hm,sizeof(int),hipMemcpyHostToDevice);
	int h_source[hm],h_dest[hm];
	for(int i=0;i<hm;i++)
    {
       scanf("%d%d",&h_source[i],&h_dest[i]);
       //h_source[i]--;
       //h_dest[i]--;
    }
    //for(int i=0;i<hm;i++)
      //  printf("%d %d\n",h_source[i]+1,h_dest[i]+1);
    int *d_source,*d_dest;
    hipMalloc((void**)&d_source,hm*sizeof(int));
    hipMalloc((void**)&d_dest,hm*sizeof(int));
    hipMemcpy(d_source,&h_source,hm*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_dest,&h_dest,hm*sizeof(int),hipMemcpyHostToDevice);
    int hs;
    scanf("%d",&hs);
    //hs--;
    int *ds;
    hipMalloc((void**)&ds,sizeof(int));
    hipMemcpy(ds,&hs,sizeof(int),hipMemcpyHostToDevice);
    int hrow=hn,hcol=ceil(hn/32.0);
    int *drow,*dcol;
    hipMalloc((void**)&drow,sizeof(int));
    hipMalloc((void**)&dcol,sizeof(int));
    hipMemcpy(drow,&hrow,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dcol,&hcol,sizeof(int),hipMemcpyHostToDevice);

    unsigned dom[hrow*hcol];
    cout<<hrow<<" "<<hcol<<endl;
    for(int i=0;i<hrow;i++)
    {
    	  if(i!=hs)
    	  {
         for(int j=0;j<hcol;j++)
       	  dom[i*hcol+j]=~(dom[i*hcol+j]&0);
        }else
        {
         for(int j=0;j<hcol;j++)
       	  dom[i*hcol+j]&=0;
        }
    }
    dom[(hs*hcol)+(hs>>5)]=(1u<<(hs&31));
    /*for(int i=0;i<hcol*hrow;i++)
    	cout<<dom[i]<<" "<<endl;*/
    unsigned *d_dom;
    hipMalloc((void**)&d_dom,hrow*hcol*sizeof(unsigned));
    hipMemcpy(d_dom,&dom,hrow*hcol*sizeof(unsigned),hipMemcpyHostToDevice);
    int cnt=0;
    clock_t tStart = clock();
    while(1)
    {
        cnt++;
        cout<<cnt<<endl;
    	bool hf=true;
        bool *df;
    	hipMalloc((void**)&df,sizeof(bool));
    	hipMemcpy(df,&hf,sizeof(bool),hipMemcpyHostToDevice);
    	fun<<<ceil(hm/512.0),512>>>(dn,dm,d_source,d_dest,ds,drow,dcol,d_dom,df);
    	hipMemcpy(&hf,df,sizeof(bool),hipMemcpyDeviceToHost);
    	if(hf)
    		break;
    }
    printf("Time taken: %.2fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
    cout<<cnt<<endl;
    hipMemcpy(dom,d_dom,hrow*hcol*sizeof(unsigned),hipMemcpyDeviceToHost);
    int a[hrow+1];
    memset(a,0,sizeof(a));
    for(int i=0;i<hrow;i++)
    {
        int ans=0;
      for(int j=0;j<hcol;j++)
      {
        if(j!=hcol-1)
        {
          int val=1;
          for(int k=0;k<32;k++)
          {
           if(dom[i*hcol+j]&val)
             ans++;
           val<<=1;
          }
        }else
        {
          int val=1;
          int temp=hrow-32*(hcol-1);
          for(int k=0;k<temp;k++)
          {
            if(dom[i*hcol+j]&val)
              ans++;
            val<<=1;
          }
        }  
      }
      a[ans]++;
    }
    for(int i=1;i<=hrow;i++)
    {
      if(a[i])
        cout<<i<<" "<<a[i]<<endl;
    }
	return 0;
}
