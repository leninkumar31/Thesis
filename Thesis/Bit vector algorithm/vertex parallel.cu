#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void fun(int *n,int *m,int *v,int *e,int *s,int *col,unsigned *dom,unsigned *temp,bool *f)
{
    int curr=blockIdx.x*blockDim.x+threadIdx.x;
    if((curr!=(*s))&&(curr<(*n)))
    {
       int c=*col;
       /*unsigned int var[MAXC];
       for(int i=0;i<c;i++)
        var[i]=dom[curr*c+i];*/
       int num=(curr==(*n-1)?(*m-v[curr]):(v[curr+1]-v[curr]));
       for(int i=0;i<num;i++)
       {
       	 int pre=e[v[curr]+i];
       	 for(int k=0;k<c;k++)
       	 {
          if(i==0)
           temp[curr*c+k]=dom[pre*c+k];
          else
         	 temp[curr*c+k]&=dom[pre*c+k];
         }
       }
       temp[(curr*c)+(curr>>5)]|=(1u<<(curr&31));
       if(*f)
       {
          for(int i=0;i<c;i++)
          {
          	if(temp[curr*c+i]!=dom[curr*c+i])
          	{
          		*f=false;
          		break;
          	}
          }
       }
    }
}
int main()
{
	freopen("input.txt","r",stdin);
  freopen("output.txt","w",stdout);
  int hn,hm;
	scanf("%d%d",&hn,&hm);
	int *dn,*dm;
    hipMalloc((void **)&dn,sizeof(int));
    hipMalloc((void **)&dm,sizeof(int));
    hipMemcpy(dn,&hn,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dm,&hm,sizeof(int),hipMemcpyHostToDevice);
    //vector<int> adj[n];
    vector<int> adjR[hn];
    for(int i=0;i<hm;i++)
    {
    	int u,v;
    	scanf("%d%d",&u,&v);
    	//u--,v--;
    	//adj[u].push_back(v);
    	adjR[v].push_back(u);
    }
    //int hv[n],he[m];
   /* int k=0;
    for(int i=0;i<n;i++)
    {
    	hv[i]=k;
    	for(int j=0;j<adj[i].size();j++)
          he[k++]=adj[i][j];
    }
    int *dv,*de;
    cudaMalloc((void**)&dv,hn*sizeof(int));
    cudaMalloc((void**)&de,hm*sizeof(int));
    cudaMemcpy(dv,&hv,hn*sizeof(int),cudaMemcpyHostToDevice);
    cudaMemcpy(de,&he,hm*sizeof(int),cudaMemcpyHostToDevice);*/
    int hvR[hn],heR[hm];
    int k=0;
    for(int i=0;i<hn;i++)
    {
    	hvR[i]=k;
    	for(int j=0;j<adjR[i].size();j++)
    		heR[k++]=adjR[i][j];
    }
   /* for(int i=0;i<hn;i++)
    	cout<<hvR[i]<<" ";
    cout<<endl;*/
    int *dvR,*deR;
    hipMalloc((void**)&dvR,hn*sizeof(int));
    hipMalloc((void**)&deR,hm*sizeof(int));
    hipMemcpy(dvR,&hvR,hn*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(deR,&heR,hm*sizeof(int),hipMemcpyHostToDevice);
    int hs;
    scanf("%d",&hs);
    //hs--;
    int *ds;
    hipMalloc((void**)&ds,sizeof(int));
    hipMemcpy(ds,&hs,sizeof(int),hipMemcpyHostToDevice);
    int row=hn,col=ceil(hn/32.0);
    int *drow,*dcol;
    hipMalloc((void **)&drow,sizeof(int));
    hipMalloc((void **)&dcol,sizeof(int));
    hipMemcpy(drow,&row,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dcol,&col,sizeof(int),hipMemcpyHostToDevice);
    unsigned dom[row*col];
    //cout<<(1u<<(hs&31))<<endl;
    for(int i=0;i<row;i++)
    {
      if(i!=hs)
      {
       for(int j=0;j<col;j++)
       	dom[i*col+j]=~(dom[i*col+j]&0);
      }else
      {
        for(int j=0;j<col;j++)
          dom[i*col+j]=(dom[i*col+j]&0);
      }
    }
    dom[(hs*col)+(hs>>5)]|=(1u<<(hs&31));
    /*for(int i=0;i<col*row;i++)
    	cout<<dom[i]<<" "<<endl;*/
    unsigned *d_dom,*temp;
    hipMalloc((void**)&d_dom,row*col*sizeof(unsigned));
    hipMalloc((void**)&temp,row*col*sizeof(unsigned));
    int cnt=0;
    clock_t tStart = clock();
    while(1)
    {
      cnt++;
    	bool hf=true;
      //cout<<cnt<<" "<<hf<<endl;
      /*for(int i=0;i<col;i++)
        cout<<dom[hs*col+i]<<" ";
      cout<<endl;*/
    	bool *df;
    	hipMalloc((void**)&df,sizeof(bool));
    	hipMemcpy(df,&hf,sizeof(bool),hipMemcpyHostToDevice);
      hipMemcpy(d_dom,dom,row*col*sizeof(unsigned),hipMemcpyHostToDevice);
      hipMemcpy(temp,dom,row*col*sizeof(unsigned),hipMemcpyHostToDevice);
    	fun<<<ceil(hn/512.0),512>>>(dn,dm,dvR,deR,ds,dcol,d_dom,temp,df);
      hipMemcpy(dom,temp,row*col*sizeof(unsigned),hipMemcpyDeviceToHost);
    	hipMemcpy(&hf,df,sizeof(bool),hipMemcpyDeviceToHost);
      //cout<<hf<<endl;
    	if(hf)
    		break;
    }
    printf("Time taken: %.2fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
    cout<<cnt<<endl;
    int a[row+1];
    memset(a,0,sizeof(a));
    for(int i=0;i<row;i++)
    {
      int ans=0;
      for(int j=0;j<col;j++)
      {
        if(j!=col-1)
        {
          unsigned int val=1;
          for(int k=0;k<32;k++)
          {
           if(dom[i*col+j]&val)
             ans++;
           val<<=1;
          }
        }else
        {
          unsigned int val=1;
          int temp=hn-32*(col-1);
          for(int k=0;k<temp;k++)
          {
            if(dom[i*col+j]&val)
              ans++;
            val<<=1;
          }
        }
      }
      if(i==hs)
        cout<<ans<<endl;
      a[ans]++;
    }
    for(int i=1;i<=row;i++)
    {
      if(a[i])
        cout<<i<<" "<<a[i]<<endl;
    }
    /*for(int i=0;i<col*row;i++)
    	cout<<dom[i]<<" "<<endl;*/
    /*for(int i=0;i<row;i++)
    {
    	for(int j=0;j<col;j++)
    		cout<<dom[i*col+j]<<" ";
    	cout<<endl;
    }*/
	return 0;
}
