#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#define MAXN 10000
using namespace std;
__global__ void pbfs(int *dn,int *dm,bool *dvis,bool *dq,int *vertex,int *dedges,bool *dflag)
{
    int u=threadIdx.x+blockDim.x*blockIdx.x;
    if(dq[u]&&u<(*dn))
    {
       dq[u]=false;
       int num=(u==(*dn-1)?(*dm-vertex[u]):(vertex[u+1]-vertex[u]));
       for(int i=0;i<num;i++)
       {
     	  int v=dedges[vertex[u]+i];
     	  if(!dvis[v])
     	  {
     		   dvis[v]=true;
     		   dq[v]=true;
     		   *dflag=true;
     	  }
       }
     }
}
vector<int> dom[MAXN],adj[MAXN];
bool vis[MAXN],temp[MAXN];
bool fvis[MAXN][MAXN];
void bfs(int s)
{
   queue<int> q;
   q.push(s);
   vis[s]=true;
   while(!q.empty())
   {
     int u=q.front();
     q.pop();
     for(int i=0;i<adj[u].size();i++)
     {
      int v=adj[u][i];
      if(!vis[v])
      {
        vis[v]=true;
        q.push(v);
      }
     }
   }
}
int main()
{
    freopen("input.txt","r",stdin);
    freopen("t1.txt","w",stdout);
    int n,m;
    cin>>n>>m;
    int *dn,*dm;
    hipMalloc((void **)&dn,sizeof(int));
    hipMalloc((void **)&dm,sizeof(int));
    hipMemcpy(dn,&n,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dm,&m,sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<m;i++)
    {
       int u,v;
       cin>>u>>v;
       //u--;
       //v--;
       adj[u].push_back(v);
    }
    int vertexoff[n],edges[m];
    int k=0;
    for(int i=0;i<n;i++)
    {
    	vertexoff[i]=k;
    	for(int j=0;j<adj[i].size();j++)
    		edges[k++]=adj[i][j];
    }
    int *dvertexoff,*dedges;
    hipMalloc((void**)&dvertexoff,n*sizeof(int));
    hipMalloc((void**)&dedges,m*sizeof(int));
    hipMemcpy(dvertexoff,&vertexoff,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dedges,&edges,m*sizeof(int),hipMemcpyHostToDevice);
    int hs;
    scanf("%d",&hs);
    //hs--;
    dom[hs].push_back(hs);
    for(int i=0;i<n;i++)
    {
        if(i!=hs)
        {
          dom[i].push_back(hs);
          dom[i].push_back(i);
        }
    }
    bfs(hs);
    //cout<<hs<<endl;
    for(int i=0;i<n;i++)
    {
    	if(!vis[i])
    		temp[i]=true;
    	else
    		temp[i]=false;
      fvis[0][i]=false;
    }
    clock_t tStart = clock();
    for(int i=1;i<n;i++)
    {
    	if(!temp[i])
    	{
    		    bool q[n];
            for(int j=0;j<n;j++)
            {
            	q[j]=false;
            	if(!temp[j])
            		vis[j]=false;
            	else
            		vis[j]=true;
            }
            q[hs]=true;
            vis[hs]=true;
            vis[i]=true;
            bool *dvis,*dq;
            hipMalloc((void**)&dvis,n*sizeof(bool));
            hipMemcpy(dvis,&vis,n*sizeof(bool),hipMemcpyHostToDevice);
            hipMalloc((void**)&dq,n*sizeof(bool));
            hipMemcpy(dq,&q,n*sizeof(bool),hipMemcpyHostToDevice);
            bool flag;
            do
            {
              flag=false;
              bool *dflag;
              hipMalloc((void**)&dflag,sizeof(bool));
              hipMemcpy(dflag,&flag,sizeof(bool),hipMemcpyHostToDevice);
    	        pbfs<<<ceil(n/512.0),512>>>(dn,dm,dvis,dq,dvertexoff,dedges,dflag);
              hipMemcpy(&flag,dflag,sizeof(bool),hipMemcpyDeviceToHost);
            }while(flag);
            hipMemcpy(fvis[i],dvis,n*sizeof(bool),hipMemcpyDeviceToHost);
            /*update<<<ceil(n/512.0),512>>>(dvis,dn);
            for(int j=0;j<n;j++)
            {
            	if(!vis[j])
            		dom[j].push_back(i);
            }*/
    	}
    }
    printf("Time taken: %.2fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
    for(int i=0;i<n;i++)
    {
        for(int j=1;j<n;j++)
        {
          if(!fvis[i][j])
            dom[j].push_back(i);
        }
    }
    for(int i=0;i<n;i++)
    {
      for(int j=0;j<dom[i].size();j++)
        cout<<dom[i][j]+1<<" ";
      cout<<endl;
    }
	return 0;
}
